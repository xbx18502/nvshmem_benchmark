/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include "coll_test.h"
#include "host/nvshmem_coll_api.h"
#include <cstdio>
#define DATATYPE int64_t

int main(int argc, char **argv) {
    int status = 0;
    int mype, npes;
    size_t size = MAX_ELEMS * 2 * sizeof(DATATYPE);
    size_t alloc_size;
    int num_elems;
    DATATYPE *buffer = NULL;
    DATATYPE *h_buffer = NULL;
    DATATYPE *d_source, *d_dest;
    DATATYPE *h_source, *h_dest;
    char size_string[100];
    uint64_t size_array[MAX_ELEMS_LOG + 1];
    double latency_array[MAX_ELEMS_LOG + 1];
    hipStream_t stream;
    int PE_root = 0;

    memset(size_array, 0, (MAX_ELEMS_LOG + 1) * sizeof(uint64_t));
    memset(latency_array, 0, (MAX_ELEMS_LOG + 1) * sizeof(double));

    DEBUG_PRINT("symmetric size requested %lu\n", size);
    sprintf(size_string, "%lu", size);

    status = setenv("NVSHMEM_SYMMETRIC_SIZE", size_string, 1);
    if (status) {
        fprintf(stderr, "setenv failed \n");
        status = -1;
        goto out;
    }

    init_wrapper(&argc, &argv);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    (void)npes;  // Silence unused variable warning
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    // num_elems = MAX_ELEMS / 2;
    num_elems = 1;
    alloc_size = num_elems * 2 * sizeof(DATATYPE);

    CUDA_CHECK(hipHostAlloc(&h_buffer, alloc_size, hipHostMallocDefault));
    h_source = (DATATYPE *)h_buffer;
    h_dest = (DATATYPE *)&h_source[num_elems];

    buffer = (DATATYPE *)nvshmem_malloc(alloc_size);
    if (!buffer) {
        fprintf(stderr, "nvshmem_malloc failed \n");
        status = -1;
        goto out;
    }
    d_source = (DATATYPE *)buffer;
    d_dest = (DATATYPE *)&d_source[num_elems];
    *h_buffer = 111;
    CUDA_CHECK(hipMemcpyAsync(d_source, h_source, 1, hipMemcpyHostToDevice, stream));                      
    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, 1,hipMemcpyHostToDevice, stream));
    nvshmem_int64_broadcast(NVSHMEM_TEAM_WORLD, d_dest, d_source, (size_t)1, PE_root);
    // if (!mype) {
    //     print_table("broadcast", "32-bit", "size (bytes)", "latency", "us", '-', size_array,
    //                 latency_array, MAX_ELEMS_LOG + 1);
    // }
    CUDA_CHECK(hipMemcpyAsync(h_source, d_source, 1,hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipMemcpyAsync(h_dest,d_dest, 1,hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    // RUN_COLL(broadcast, BCAST, int64, int64_t, d_source, h_source, d_dest, h_dest, npes, PE_root,
    //          stream, size_array, latency_array);
    // if (!mype) {
    //     print_table("broadcast", "64-bit", "size (bytes)", "latency", "us", '-', size_array,
    //                 latency_array, MAX_ELEMS_LOG + 1);
    // }
    nvshmem_barrier_all();  
    printf("h_dest = %ld\n", *h_dest);
    CUDA_CHECK(hipHostFree(h_buffer));
    nvshmem_free(buffer);

    nvshmem_barrier_all();

    CUDA_CHECK(hipStreamDestroy(stream));

    finalize_wrapper();

out:
    return 0;
}
