#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"
#include <iostream>
#include <assert.h>
#define MAX_MSG_SIZE (32 * 1024 * 1024)

#define MAX_ITERS 200
#define MAX_SKIP 20
#define BLOCKS 4
#define THREADS_PER_BLOCK 1024

#define CUDA_CHECK(stmt)                                  \
do {                                                      \
    hipError_t result = (stmt);                          \
    if (hipSuccess != result) {                          \
        fprintf(stderr, "[%s:%d] CUDA failed with %s \n", \
         __FILE__, __LINE__, hipGetErrorString(result)); \
        exit(-1);                                         \
    }                                                     \
} while (0)

const int message_size = 1<<22;

int skip = 1;
int loop = 1;
int skip_large = 0;
int loop_large = 100;
int large_message_size = 8192;

__global__ void simple_shift(double *destination, int size) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    // nvshmem_int_p(destination, mype, peer);
    nvshmemx_double_put_nbi_block(destination,destination, size, peer);
}

__global__ void bw(double* dest, int size){
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;
    nvshmemx_double_put_nbi_block(dest + (bid * (size / nblocks)),
                                  dest + (bid * (size / nblocks)), size / nblocks, peer);

}

__global__ void bw2(double *data_d, volatile unsigned int *counter_d, int len, int pe, int iter) {
    int i, peer;
    unsigned int counter;
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;

    peer = !pe;
    for (i = 0; i < iter; i++) {
        nvshmemx_double_put_nbi_block(data_d + (bid * (len / nblocks)),
                                      data_d + (bid * (len / nblocks)), len / nblocks, peer);

        // synchronizing across blocks
        __syncthreads();
        if (!tid) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }
        __syncthreads();
    }

    // synchronize and call nvshme_quiet
    __syncthreads();
    if (!tid) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
    __syncthreads();
}
int main (int argc, char *argv[]) {
    int mype_node;
    double* msg = (double*)malloc(sizeof(double)*message_size);
    double* msg_main = (double*)malloc(sizeof(double)*message_size);
    hipStream_t stream;
    int rank, nranks;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;
    std::cout<<"complete nvshmemx_init_attr_t attr"<<std::endl;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    std::cout<<"complete MPI_Init"<<std::endl;
    attr.mpi_comm = &mpi_comm;
    std::cout<<"complete attr.mpi_comm = &mpi_comm"<<std::endl;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    std::cout<<"complete nvshmemx_init_attr"<<std::endl;
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    std::cout<<"complete nvshmemx_init_attr"<<std::endl;
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    std::cout<<"complete hipSetDevice"<<std::endl;
    double *destination = (double *) nvshmem_malloc (sizeof(double)*message_size);
    std::cout<<"complete nvshmem_malloc"<<std::endl;
    // CUDA_CHECK(hipMemset(destination, 0, sizeof(int)*message_size));
    for(int i=0; i<message_size; i++) {
        msg_main[i] = 42;
    }
    CUDA_CHECK(hipMemcpy(destination, msg_main, sizeof(double)*message_size,
                hipMemcpyHostToDevice));
    std::cout<<"complete *destination init"<<std::endl;
    #define MAX_MSG_SIZE_PT2PT (1 << 20)
    #define FLOAT_PRECISION 2
    long long size;
        #define HEADER "# " "OSU OpenSHMEM Put Bandwidth Test" " v" "7.5" "\n"
    #define FIELD_WIDTH 18
    if (0 == mype_node) {
        fprintf(stdout, HEADER);
        fprintf(stdout, "%-*s%*s\n", 10, "# Size", FIELD_WIDTH,
                "Bandwidth (MB/s)");
        fflush(stdout);
    }
    hipEvent_t start, stop;
    hipEvent_t start0, stop0;
    hipEvent_t start1, stop1;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventCreate(&start0);
    hipEventCreate(&stop0);
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    std::cout<<"complete hipEventCreate"<<std::endl;
    unsigned int *counter_d;
    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    std::cout<<"init counter_d"<<std::endl;

    for (size = 1; size <= MAX_MSG_SIZE_PT2PT; size = (size ? size * 2 : 1)) {
        nvshmem_barrier_all();
        // CUDA_CHECK(hipDeviceSynchronize());
        // nvshmem_barrier_all();
        // nvshmem_sync_all();
        if (0 == mype_node) {
            for (int i = 0; i < skip; i++) {
                CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
                bw2<<<4,1024>>> (destination, counter_d, size, mype_node,1);
                
            }
            // nvshmem_barrier_all();
            
        }
        nvshmem_barrier_all();
        hipEventRecord(start);
        CUDA_CHECK(hipEventSynchronize(start));
        if(0==mype_node){
            hipEventRecord(start0);
            CUDA_CHECK(hipEventSynchronize(start0));
        }
        if(1==mype_node){
            hipEventRecord(start1);
            CUDA_CHECK(hipEventSynchronize(start1));
        }
        
        
        if (0 == mype_node) {
            for (int i = 0; i < loop; i++) {
                CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
                bw2<<<4,1024>>> (destination, counter_d, size, mype_node,1);
                
            }
            // nvshmem_barrier_all();
            
        }
        else{
            // nvshmem_barrier_all();
        }
        // nvshmem_sync_all();
        nvshmem_barrier_all();
        hipEventRecord(stop);
        CUDA_CHECK(hipEventSynchronize(stop));
        if(0==mype_node){
            hipEventRecord(stop0);
            CUDA_CHECK(hipEventSynchronize(stop0));
        }
        if(1==mype_node){
            hipEventRecord(stop1);
            CUDA_CHECK(hipEventSynchronize(stop1));
        }
        
        

        //nvshmem_barrier_all();
        //CUDA_CHECK(hipStreamSynchronize(stream));
        //nvshmem_barrier_all();
        double mb_total = 0.0;
        double t_total = 0.0;
        float milliseconds = 0.0;
        float milliseconds0 = 0.0;
        float milliseconds1 = 0.0;
        if (0 == mype_node) {
            mb_total = size * loop *8/ ( 1e6);
            hipEventElapsedTime(&milliseconds, start, stop);
            t_total = milliseconds/1e3;
            double bw = mb_total / t_total;
            fprintf(stdout, "%-*d%*.*f\n", 10, size*8, FIELD_WIDTH,
                    FLOAT_PRECISION, bw);
            // hipEventElapsedTime(&milliseconds0, start0, stop0);
            // fprintf(stdout, "mype0 elaped time = %f\n", milliseconds0);
            // fprintf(stdout, "mype0 start time = %f\n", start);
            // fprintf(stdout, "mype0 stop time = %f\n", stop);
            fflush(stdout);
            //std::cout<<"PE0 finish print"<<std::endl;
            nvshmem_barrier_all();
        }
        else{
            // hipEventElapsedTime(&milliseconds1, start1, stop1);
            // fprintf(stdout, "mype1 elaped time = %f\n", milliseconds1);
            std::cout<<"PE1 finish print"<<std::endl;
            nvshmem_barrier_all();
        }
    }
    std::cout<<"finish the loop"<<std::endl;
    hipEventDestroy(start);
    std::cout<<"finish destroy the start"<<std::endl;
    hipEventDestroy(stop);
    std::cout<<"finish destroy the stop"<<std::endl;
    CUDA_CHECK(hipStreamSynchronize(stream));
    nvshmem_free(destination);
    std::cout<<"finish nvshmem_free"<<std::endl;
    nvshmem_finalize();
    std::cout<<"finish nvshmem_finalize"<<std::endl;
    CUDA_CHECK(hipStreamDestroy(stream));
    std::cout<<"finish destroy the stream"<<std::endl;
    MPI_Finalize();
    std::cout<<"finish MPI_Finalize"<<std::endl;
    return 0;
}
