#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "nvshmem.h"
#include "nvshmemx.h"

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

#define THREADS_PER_BLOCK 1024

__global__ void set_and_shift_kernel(float *send_data, float *recv_data, int num_elems, int mype,
                                     int npes) {
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    /* set the corresponding element of send_data */
    if (thread_idx < num_elems) send_data[thread_idx] = mype;

    int peer = (mype + 1) % npes;
    /* Every thread in block 0 calls nvshmemx_float_put_block. Alternatively,
       every thread can call shmem_float_p, but shmem_float_p has a disadvantage
       that when the destination GPU is connected via IB, there will be one rma
       message for every single element which can be detrimental to performance.
       And the disadvantage with shmem_float_put is that when the destination GPU is p2p
       connected, it cannot leverage multiple threads to copy the data to the destination
       GPU. */
    int block_offset = blockIdx.x * blockDim.x;
    nvshmemx_float_put_block(recv_data + block_offset, send_data + block_offset,
                             min(blockDim.x, num_elems - block_offset),
                             peer); /* All threads in a block call the API
                                       with the same arguments */
}

int main(int c, char *v[]) {
    int mype, npes, mype_node;
    float *send_data, *recv_data;
    int num_elems = 8192;
    int num_blocks;

    nvshmem_init();

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    // application picks the device each PE will use
    CUDA_CHECK(hipSetDevice(mype_node));
    send_data = (float *)nvshmem_malloc(sizeof(float) * num_elems);
    recv_data = (float *)nvshmem_malloc(sizeof(float) * num_elems);
    assert(send_data != NULL && recv_data != NULL);

    assert(num_elems % THREADS_PER_BLOCK == 0); /* for simplicity */
    num_blocks = num_elems / THREADS_PER_BLOCK;

    set_and_shift_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(send_data, recv_data, num_elems, mype,
                                                            npes);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    /* Do data validation */
    float *host = new float[num_elems];
    CUDA_CHECK(hipMemcpy(host, recv_data, num_elems * sizeof(float), hipMemcpyDefault));
    int ref = (mype - 1 + npes) % npes;
    bool success = true;
    for (int i = 0; i < num_elems; ++i) {
        if (host[i] != ref) {
            printf("Error at %d of rank %d: %f\n", i, mype, host[i]);
            success = false;
            break;
        }
    }

    if (success) {
        printf("[%d of %d] run complete \n", mype, npes);
    } else {
        printf("[%d of %d] run failure \n", mype, npes);
    }

    nvshmem_free(send_data);
    nvshmem_free(recv_data);

    nvshmem_finalize();

    return 0;
}