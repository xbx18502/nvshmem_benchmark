#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"
#include <iostream>
#include <assert.h>
#define MAX_MSG_SIZE (32 * 1024 * 1024)

#define MAX_ITERS 200
#define MAX_SKIP 20
#define BLOCKS 4
#define THREADS_PER_BLOCK 1024

#define CUDA_CHECK(stmt)                                  \
do {                                                      \
    hipError_t result = (stmt);                          \
    if (hipSuccess != result) {                          \
        fprintf(stderr, "[%s:%d] CUDA failed with %s \n", \
         __FILE__, __LINE__, hipGetErrorString(result)); \
        exit(-1);                                         \
    }                                                     \
} while (0)

const int message_size = 1<<22;

int skip = 1000;
int loop = 10000;
int skip_large = 10;
int loop_large = 100;
int large_message_size = 8192;

__global__ void simple_shift(double *destination, int size) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    // nvshmem_int_p(destination, mype, peer);
    nvshmemx_double_put_nbi_block(destination,destination, size, peer);
}

__global__ void bw(double* dest, int size){
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;
    nvshmemx_double_put_nbi_block(dest + (bid * (size / nblocks)),
                                  dest + (bid * (size / nblocks)), size / nblocks, peer);

}

__global__ void bw2(double *data_d, volatile unsigned int *counter_d, int len, int pe, int iter) {
    int i, peer;
    unsigned int counter;
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;

    peer = !pe;
    for (i = 0; i < iter; i++) {
        nvshmemx_double_put_nbi_block(data_d + (bid * (len / nblocks)),
                                      data_d + (bid * (len / nblocks)), len / nblocks, peer);

        // synchronizing across blocks
        __syncthreads();
        if (!tid) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }
        __syncthreads();
    }

    // synchronize and call nvshme_quiet
    __syncthreads();
    if (!tid) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
    __syncthreads();
}
int main (int argc, char *argv[]) {
    int mype_node;
    double* msg = (double*)malloc(sizeof(double)*message_size);
    double* msg_main = (double*)malloc(sizeof(double)*message_size);
    hipStream_t stream;
    int rank, nranks;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;
    std::cout<<"complete nvshmemx_init_attr_t attr"<<std::endl;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    std::cout<<"complete MPI_Init"<<std::endl;
    attr.mpi_comm = &mpi_comm;
    std::cout<<"complete attr.mpi_comm = &mpi_comm"<<std::endl;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    std::cout<<"complete nvshmemx_init_attr"<<std::endl;
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    std::cout<<"complete nvshmemx_init_attr"<<std::endl;
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    std::cout<<"complete hipSetDevice"<<std::endl;
    double *destination = (double *) nvshmem_malloc (sizeof(double)*message_size);
    std::cout<<"complete nvshmem_malloc"<<std::endl;
    // CUDA_CHECK(hipMemset(destination, 0, sizeof(int)*message_size));
    for(int i=0; i<message_size; i++) {
        msg_main[i] = 42;
    }
    CUDA_CHECK(hipMemcpy(destination, msg_main, sizeof(double)*message_size,
                hipMemcpyHostToDevice));
    std::cout<<"complete *destination init"<<std::endl;
    #define MAX_MSG_SIZE_PT2PT (1 << 20)
    #define FLOAT_PRECISION 2
    int size;
        #define HEADER "# " "OSU OpenSHMEM Put Bandwidth Test" " v" "7.5" "\n"
    #define FIELD_WIDTH 18
    if (0 == mype_node) {
        fprintf(stdout, HEADER);
        fprintf(stdout, "%-*s%*s\n", 10, "# Size", FIELD_WIDTH,
                "Bandwidth (MB/s)");
        fflush(stdout);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    std::cout<<"complete hipEventCreate"<<std::endl;
    for (size = 1; size <= MAX_MSG_SIZE_PT2PT; size = (size ? size * 2 : 1)) {
        if (size > large_message_size) {
            loop = loop_large = 100;
            skip = skip_large = 0;
        }
        nvshmemx_barrier_all_on_stream(stream);
        if (0 == mype_node) {
            for (int i = 0; i < loop + skip; i++) {
                if (i == skip) {
                    hipEventRecord(start);
                }

                //simple_shift<<<1, 1 , 0, stream>>>(destination, size);
                //CUDA_CHECK(hipStreamSynchronize(stream)); 
                // bw<<<4, 1024, 0, stream>>>(destination, size);
                bw<<<4, 1024, 0, stream>>>(destination, size);
                CUDA_CHECK(hipStreamSynchronize(stream));
            }

            hipEventRecord(stop);
            CUDA_CHECK(hipEventSynchronize(stop));
        }
        // std::cout<<"start = "<<start<<std::endl;
        // std::cout<<"stop = "<<stop<<std::endl;
        nvshmemx_barrier_all_on_stream(stream);
        double mb_total = 0.0;
        double t_total = 0.0;
        float milliseconds = 0.0;
        if (0 == mype_node) {
            mb_total = size * loop *8/ ( 1e6);
            // std::cout<<"mb_total = "<<mb_total<<std::endl;
            hipEventElapsedTime(&milliseconds, start, stop);
            t_total = milliseconds/1e3;
            //std::cout<<"t_total = "<<t_total<<std::endl;
            double bw = mb_total / t_total;
            //std::cout<<"bw = "<<bw<<std::endl;
            fprintf(stdout, "%-*d%*.*f\n", 10, size, FIELD_WIDTH,
                    FLOAT_PRECISION, bw);
            fflush(stdout);
        }
        else{
            /*
            CUDA_CHECK(hipMemcpyAsync(msg, destination, sizeof(double)*message_size,
            hipMemcpyDeviceToHost, stream));
            std::cout<<"complete hipMemcpyAsync"<<std::endl;
            CUDA_CHECK(hipStreamSynchronize(stream));
            std::cout<<"complete hipStreamSynchronize"<<std::endl;
            printf("%d: received message ", nvshmem_my_pe());
            for(int i=0; i<32; i++) {
                printf("%f ", msg[i]);
            }
            printf("\n");
            */
        }
    }
    // simple_shift<<<1, 1, 0, stream>>>(destination);
    // std::cout<<"complete simple_shift"<<std::endl;
    // nvshmemx_barrier_all_on_stream(stream);
    // std::cout<<"complete nvshmemx_barrier_all_on_stream"<<std::endl;
    // CUDA_CHECK(hipMemcpyAsync(msg, destination, sizeof(double)*message_size,
    //             hipMemcpyDeviceToHost, stream));
    // std::cout<<"complete hipMemcpyAsync"<<std::endl;
    // CUDA_CHECK(hipStreamSynchronize(stream));
    // std::cout<<"complete hipStreamSynchronize"<<std::endl;
    // printf("%d: received message ", nvshmem_my_pe());
    // for(int i=0; i<message_size; i++) {
    //     printf("%f ", msg[i]);
    // }
    // printf("\n");
    CUDA_CHECK(hipStreamSynchronize(stream));
    nvshmem_free(destination);
    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}
